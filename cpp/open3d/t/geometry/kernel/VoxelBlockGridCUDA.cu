#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include "open3d/core/Dispatch.h"
#include "open3d/core/Dtype.h"
#include "open3d/core/MemoryManager.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/SizeVector.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/hashmap/CUDA/StdGPUHashBackend.h"
#include "open3d/core/hashmap/DeviceHashBackend.h"
#include "open3d/core/hashmap/Dispatch.h"
#include "open3d/core/hashmap/HashMap.h"
#include "open3d/t/geometry/kernel/GeometryIndexer.h"
#include "open3d/t/geometry/kernel/GeometryMacros.h"
#include "open3d/t/geometry/kernel/VoxelBlockGrid.h"
#include "open3d/t/geometry/kernel/VoxelBlockGridImpl.h"
#include "open3d/utility/Logging.h"

namespace open3d {
namespace t {
namespace geometry {
namespace kernel {
namespace voxel_grid {

struct Coord3i {
    OPEN3D_HOST_DEVICE Coord3i(index_t x, index_t y, index_t z)
        : x_(x), y_(y), z_(z) {}
    OPEN3D_HOST_DEVICE bool operator==(const Coord3i &other) const {
        return x_ == other.x_ && y_ == other.y_ && z_ == other.z_;
    }

    index_t x_;
    index_t y_;
    index_t z_;
};

struct Coord3iHash {
    size_t operator()(const Coord3i &k) const {
        static const size_t p0 = 73856093;
        static const size_t p1 = 19349669;
        static const size_t p2 = 83492791;

        return (static_cast<size_t>(k.x_) * p0) ^
               (static_cast<size_t>(k.y_) * p1) ^
               (static_cast<size_t>(k.z_) * p2);
    }
};


template <typename src_t, typename dst_t>
static void OPEN3D_DEVICE CUDAEqElementKernel(const void* lhs,
                                                   const void* rhs,
                                                   void* dst) {
    *static_cast<dst_t*>(dst) = static_cast<dst_t>(
            *static_cast<const src_t*>(lhs) == *static_cast<const src_t*>(rhs));
}


void PointCloudRayMarchingCUDA(std::shared_ptr<core::HashMap>
                &hashmap,
        const core::Tensor &points,
        const core::Tensor &pcd_normals,
        const core::Tensor &extrinsic,
        core::Tensor &voxel_block_coords,
		core::Tensor &block_pcd_coords,
        core::Tensor &block_pcd_normals,
        index_t voxel_grid_resolution,
        float voxel_size,
		index_t step_size,
		index_t tangential_step_size,
        float sdf_trunc){

        core::Device device = points.GetDevice();
        // sensor origin
        core::Tensor pose = t::geometry::InverseTransformation(extrinsic);
        index_t resolution = voxel_grid_resolution;
        float block_size = voxel_size * resolution;

        index_t n = points.GetLength();
        const float *pcd_ptr = static_cast<const float *>(points.GetDataPtr());
        const float *pcd_normals_ptr = static_cast<const float *>(pcd_normals.GetDataPtr());
        const float *origin_ptr = static_cast<const float *>(pose.GetDataPtr());
        float x_o = origin_ptr[0*4+3];
        float y_o = origin_ptr[1*4+3];
        float z_o = origin_ptr[2*4+3];

		const index_t num_blocks = tangential_step_size * tangential_step_size * 4;
        const index_t est_multipler_factor = (step_size + 1) *  num_blocks;

        core::Tensor block_coordi({est_multipler_factor * n, 3}, core::Int32, device);
    	index_t *block_coordi_ptr = static_cast<index_t *>(block_coordi.GetDataPtr());
    
		// save the block - pcd_index association [before hashing]
		core::Tensor block_pcd_lookup({est_multipler_factor * n, 1}, core::Int32, device);
    	index_t *block_pcd_lookup_ptr =
            static_cast<index_t *>(block_pcd_lookup.GetDataPtr());

        core::Tensor block_angle_lookup({est_multipler_factor * n, 1}, core::Float32, device);
    	float *block_angle_lookup_ptr =
            static_cast<float *>(block_angle_lookup.GetDataPtr());

        core::Tensor count(std::vector<index_t>{0}, {}, core::Int32, device);
    	index_t *count_ptr = static_cast<index_t *>(count.GetDataPtr());

         // populate neighbor points for association
        const float tangential_step = voxel_size;

        core::Tensor neighbor_pts_host({num_blocks, 3}, core::Dtype::Float32, core::Device("CPU:0"));
        float *neighbor_pts_host_ptr = static_cast<float *>(neighbor_pts_host.GetDataPtr());

		index_t cnt = 0;
		for (auto ii=-tangential_step_size;ii<tangential_step_size;ii++) {
            for (auto jj=-tangential_step_size;jj<tangential_step_size;jj++) {	
				neighbor_pts_host_ptr[cnt * 3 + 0] = ii*tangential_step;
                neighbor_pts_host_ptr[cnt * 3 + 1] = jj*tangential_step;
                neighbor_pts_host_ptr[cnt * 3 + 2] = 0;
				cnt++;
            }
        }

		// move from cpu host to gpu device
		core::Tensor neighbor_pts = neighbor_pts_host.To(device);
        float *neighbor_pts_ptr = static_cast<float *>(neighbor_pts.GetDataPtr());

        // for each xyz point
        core::ParallelFor(hashmap->GetDevice(), n,
                      [=] OPEN3D_DEVICE(index_t workload_idx) {
			// point in map frame
			float x = pcd_ptr[3 * workload_idx + 0];
			float y = pcd_ptr[3 * workload_idx + 1];
			float z = pcd_ptr[3 * workload_idx + 2];
			

			// Marching Ray Direction
			float x_d = x - x_o, y_d = y - y_o, z_d = z - z_o;
			float d = sqrtf(x_d * x_d + y_d * y_d + z_d * z_d);

			// unit_normal
			float unit_normal_x = x_d / d,
				unit_normal_y = y_d / d,
				unit_normal_z = z_d / d;
			float denominator = std::sqrt(unit_normal_x * unit_normal_x
										+ unit_normal_y * unit_normal_y);

			float fraction_x = unit_normal_x / denominator,
				fraction_y = unit_normal_y / denominator;

			const float t_min = (d - sdf_trunc) / d; //max(d - sdf_trunc, 0.0f) / d;
			const float t_max = (d + sdf_trunc) /  d ; // min(d + sdf_trunc, depth_max) / d;
			const float t_step = (t_max - t_min) / step_size;

			float t = t_min;

			for (index_t step = 0; step <= step_size; ++step) {

				float x_f = x_o + t * x_d;
				float y_f = y_o + t * y_d;
				float z_f = z_o + t * z_d;

				for (index_t ii = 0; ii<num_blocks; ii++){
					float x_g = 0, y_g = 0, z_g = 0;
					float x_in = neighbor_pts_ptr[ii*3 + 0], 
						y_in = neighbor_pts_ptr[ii*3 + 1],
						z_in = neighbor_pts_ptr[ii*3 + 2];
					
					x_g = x_in * fraction_y + y_in * (-fraction_x) + z_in * 0;
					y_g = x_in * (fraction_x * unit_normal_z)  
						+ y_in * (fraction_y * unit_normal_z) +
						z_in * (-denominator);
					z_g = x_in * fraction_x + y_in * unit_normal_y +
						z_in * unit_normal_z;

					index_t x_neighbor = static_cast<index_t>(
						std::floor((x_f + x_g) / block_size));
					index_t y_neighbor = static_cast<index_t>(
						std::floor((y_f + y_g) / block_size));
					index_t z_neighbor = static_cast<index_t>(
						std::floor((z_f + z_g) / block_size));
					index_t idx = atomicAdd(count_ptr, 1);
					
					// saving block coordi before hasing
					block_coordi_ptr[3 * idx + 0] = x_neighbor;
					block_coordi_ptr[3 * idx + 1] = y_neighbor;
					block_coordi_ptr[3 * idx + 2] = z_neighbor;

                    // change herustics to using distance
                    float block_pcd_dist_x = static_cast<float>(x_neighbor) - x, 
						block_pcd_dist_y = static_cast<float>(y_neighbor) - y,
						block_pcd_dist_z = static_cast<float>(z_neighbor) - z;
                    float current_distance =  std::sqrt(block_pcd_dist_x * block_pcd_dist_x
                                            + block_pcd_dist_y * block_pcd_dist_y
                                            + block_pcd_dist_z * block_pcd_dist_z);
					block_pcd_lookup_ptr[idx] = workload_idx;
					block_angle_lookup_ptr[idx] = current_distance;
		
				}
				t += t_step;
			}
		});

		index_t total_block_count = count.Item<index_t>();

		if (total_block_count == 0) {
			utility::LogError(
					"[CUDATSDFTouchKernel] No block is touched in TSDF volume, "
					"abort integration. Please check specified parameters, "
					"especially depth_scale and voxel_size");
		}
        // Step 1: Activation / Insertion into hashmap
		block_coordi = block_coordi.Slice(0, 0, total_block_count);
		core::Tensor block_buf_indices, block_masks;
		hashmap->Activate(block_coordi, block_buf_indices, block_masks);
        index_t num_unique_blocks = hashmap->Size();

		voxel_block_coords = block_coordi.IndexGet({block_masks});
        block_pcd_coords = core::Tensor(voxel_block_coords.GetShape(), core::Float32, device);
        float *block_pcd_coords_ptr = static_cast<float *>(block_pcd_coords.GetDataPtr());
        
        block_pcd_normals = core::Tensor(voxel_block_coords.GetShape(), core::Float32, device); 
        float *block_pcd_normals_ptr = static_cast<float *>(block_pcd_normals.GetDataPtr());
        // Step 2: All keys shall reside in the hashmap
        // to get depulicated key location in the hashmap
        hashmap->Find(block_coordi, block_buf_indices, block_masks);
        // assert
        if (!block_masks.All()){
            utility::LogError(
					"[PointCloudRayMarchingCUDA] Hashmap find has missing keys.");
        }
        if (block_buf_indices.Max({0}, true).Item<index_t>() != num_unique_blocks-1 ){
            utility::LogError(
					"[PointCloudRayMarchingCUDA] block_buf_indices size incorrect.");
        }
        if (num_unique_blocks != voxel_block_coords.GetShape(0)){
            utility::LogError(
					"[PointCloudRayMarchingCUDA] hashmap size incorrect.");
        }

        // Init tensor to be updated and written into hashmap value
        std::vector<float> vec_val(num_unique_blocks, 1000.f);
        std::vector<index_t> vec_index(num_unique_blocks, 0);

        core::Tensor result_value(vec_val, {num_unique_blocks, 1}, core::Float32, device);
        float *result_value_ptr = static_cast<float *>(result_value.GetDataPtr());

        core::Tensor result_index(vec_index, {num_unique_blocks, 1}, core::Int32, device);
        index_t *result_index_ptr = static_cast<index_t *>(result_index.GetDataPtr());

        index_t *block_buf_indices_ptr = static_cast<index_t *>(block_buf_indices.GetDataPtr());

        // Pass 1: do elementwise max operation for angle herustics
         core::ParallelFor(hashmap->GetDevice(), total_block_count,
                      [=] OPEN3D_DEVICE(index_t workload_idx){
            index_t hash_idx = block_buf_indices_ptr[workload_idx];
            atomicMinf(&result_value_ptr[hash_idx], block_angle_lookup_ptr[workload_idx]);
            });

        // Pass 2: find the argmax angle to pcd
        core::ParallelFor(hashmap->GetDevice(), total_block_count,
                      [=] OPEN3D_DEVICE(index_t workload_idx){
            index_t hash_idx = block_buf_indices_ptr[workload_idx];
            // finding the index with the largest angle with some precision tolerance
            if (result_value_ptr[hash_idx] > block_angle_lookup_ptr[workload_idx] - 1e-6){
                result_index_ptr[hash_idx] = block_pcd_lookup_ptr[workload_idx];
            }
            });

        // Pass 3: prepare results tensors for association of pcd index
        core::Tensor voxel_block_buf_indices, voxel_block_masks;
        hashmap->Find(voxel_block_coords, voxel_block_buf_indices, voxel_block_masks);

        index_t *voxel_block_buf_indices_ptr = static_cast<index_t *>(voxel_block_buf_indices.GetDataPtr());
        core::ParallelFor(hashmap->GetDevice(), voxel_block_coords.GetShape(0),
                      [=] OPEN3D_DEVICE(index_t workload_idx){

            index_t hash_idx = voxel_block_buf_indices_ptr[workload_idx];
            index_t pcd_idx = result_index_ptr[hash_idx];
            float pcdX = pcd_ptr[3 * pcd_idx + 0];
            float pcdY = pcd_ptr[3 * pcd_idx + 1];
            float pcdZ = pcd_ptr[3 * pcd_idx + 2];

            block_pcd_coords_ptr[3 * workload_idx + 0] = pcdX;
            block_pcd_coords_ptr[3 * workload_idx + 1] = pcdY;
            block_pcd_coords_ptr[3 * workload_idx + 2] = pcdZ;

            block_pcd_normals_ptr[3 * workload_idx + 0] = pcd_normals_ptr[3 * pcd_idx + 0];
            block_pcd_normals_ptr[3 * workload_idx + 1] = pcd_normals_ptr[3 * pcd_idx + 1];
            block_pcd_normals_ptr[3 * workload_idx + 2] = pcd_normals_ptr[3 * pcd_idx + 2];

            });
}

void PointCloudTouchCUDA(std::shared_ptr<core::HashMap> &hashmap,
                         const core::Tensor &points,
                         core::Tensor &voxel_block_coords,
                         index_t voxel_grid_resolution,
                         float voxel_size,
                         float sdf_trunc) {
    index_t resolution = voxel_grid_resolution;
    float block_size = voxel_size * resolution;

    index_t n = points.GetLength();
    const float *pcd_ptr = static_cast<const float *>(points.GetDataPtr());

    core::Device device = points.GetDevice();
    core::Tensor block_coordi({8 * n, 3}, core::Int32, device);
    index_t *block_coordi_ptr =
            static_cast<index_t *>(block_coordi.GetDataPtr());
    core::Tensor count(std::vector<index_t>{0}, {}, core::Int32, device);
    index_t *count_ptr = static_cast<index_t *>(count.GetDataPtr());

    core::ParallelFor(hashmap->GetDevice(), n,
                      [=] OPEN3D_DEVICE(index_t workload_idx) {
                          float x = pcd_ptr[3 * workload_idx + 0];
                          float y = pcd_ptr[3 * workload_idx + 1];
                          float z = pcd_ptr[3 * workload_idx + 2];

                          index_t xb_lo = static_cast<index_t>(
                                  floorf((x - sdf_trunc) / block_size));
                          index_t xb_hi = static_cast<index_t>(
                                  floorf((x + sdf_trunc) / block_size));
                          index_t yb_lo = static_cast<index_t>(
                                  floorf((y - sdf_trunc) / block_size));
                          index_t yb_hi = static_cast<index_t>(
                                  floorf((y + sdf_trunc) / block_size));
                          index_t zb_lo = static_cast<index_t>(
                                  floorf((z - sdf_trunc) / block_size));
                          index_t zb_hi = static_cast<index_t>(
                                  floorf((z + sdf_trunc) / block_size));

                          for (index_t xb = xb_lo; xb <= xb_hi; ++xb) {
                              for (index_t yb = yb_lo; yb <= yb_hi; ++yb) {
                                  for (index_t zb = zb_lo; zb <= zb_hi; ++zb) {
                                      index_t idx = atomicAdd(count_ptr, 1);
                                      block_coordi_ptr[3 * idx + 0] = xb;
                                      block_coordi_ptr[3 * idx + 1] = yb;
                                      block_coordi_ptr[3 * idx + 2] = zb;
                                  }
                              }
                          }
                      });

    index_t total_block_count = count.Item<index_t>();

    if (total_block_count == 0) {
        utility::LogError(
                "[CUDATSDFTouchKernel] No block is touched in TSDF volume, "
                "abort integration. Please check specified parameters, "
                "especially depth_scale and voxel_size");
    }
    block_coordi = block_coordi.Slice(0, 0, total_block_count);
    core::Tensor block_buf_indices, block_masks;
    hashmap->Activate(block_coordi.Slice(0, 0, count.Item<index_t>()),
                      block_buf_indices, block_masks);
    voxel_block_coords = block_coordi.IndexGet({block_masks});
}

void DepthTouchCUDA(std::shared_ptr<core::HashMap> &hashmap,
                    const core::Tensor &depth,
                    const core::Tensor &intrinsic,
                    const core::Tensor &extrinsic,
                    core::Tensor &voxel_block_coords,
                    index_t voxel_grid_resolution,
                    float voxel_size,
                    float sdf_trunc,
                    float depth_scale,
                    float depth_max,
                    index_t stride) {
    core::Device device = depth.GetDevice();
    NDArrayIndexer depth_indexer(depth, 2);
    core::Tensor pose = t::geometry::InverseTransformation(extrinsic);
    TransformIndexer ti(intrinsic, pose, 1.0f);

    // Output
    index_t rows_strided = depth_indexer.GetShape(0) / stride;
    index_t cols_strided = depth_indexer.GetShape(1) / stride;
    index_t n = rows_strided * cols_strided;

    const index_t step_size = 3;
    const index_t est_multipler_factor = (step_size + 1);

    static core::Tensor block_coordi;
    if (block_coordi.GetLength() != est_multipler_factor * n) {
        block_coordi = core::Tensor({est_multipler_factor * n, 3},
                                    core::Dtype::Int32, device);
    }

    // Counter
    core::Tensor count(std::vector<index_t>{0}, {1}, core::Dtype::Int32,
                       device);
    index_t *count_ptr = count.GetDataPtr<index_t>();
    index_t *block_coordi_ptr = block_coordi.GetDataPtr<index_t>();

    index_t resolution = voxel_grid_resolution;
    float block_size = voxel_size * resolution;
    DISPATCH_DTYPE_TO_TEMPLATE(depth.GetDtype(), [&]() {
        core::ParallelFor(device, n, [=] OPEN3D_DEVICE(index_t workload_idx) {
            index_t y = (workload_idx / cols_strided) * stride;
            index_t x = (workload_idx % cols_strided) * stride;

            float d = *depth_indexer.GetDataPtr<scalar_t>(x, y) / depth_scale;
            if (d > 0 && d < depth_max) {
                float x_c = 0, y_c = 0, z_c = 0;
                ti.Unproject(static_cast<float>(x), static_cast<float>(y), 1.0,
                             &x_c, &y_c, &z_c);
                float x_g = 0, y_g = 0, z_g = 0;
                ti.RigidTransform(x_c, y_c, z_c, &x_g, &y_g, &z_g);

                // Origin
                float x_o = 0, y_o = 0, z_o = 0;
                ti.GetCameraPosition(&x_o, &y_o, &z_o);

                // Direction
                float x_d = x_g - x_o;
                float y_d = y_g - y_o;
                float z_d = z_g - z_o;

                const float t_min = max(d - sdf_trunc, 0.0);
                const float t_max = min(d + sdf_trunc, depth_max);
                const float t_step = (t_max - t_min) / step_size;

                float t = t_min;
                index_t idx = OPEN3D_ATOMIC_ADD(count_ptr, (step_size + 1));
                for (index_t step = 0; step <= step_size; ++step) {
                    index_t offset = (step + idx) * 3;

                    index_t xb = static_cast<index_t>(
                            floorf((x_o + t * x_d) / block_size));
                    index_t yb = static_cast<index_t>(
                            floorf((y_o + t * y_d) / block_size));
                    index_t zb = static_cast<index_t>(
                            floorf((z_o + t * z_d) / block_size));

                    block_coordi_ptr[offset + 0] = xb;
                    block_coordi_ptr[offset + 1] = yb;
                    block_coordi_ptr[offset + 2] = zb;

                    t += t_step;
                }
            }
        });
    });

    index_t total_block_count = static_cast<index_t>(count[0].Item<index_t>());
    if (total_block_count == 0) {
        utility::LogError(
                "No block is touched in TSDF volume, "
                "abort integration. Please check specified parameters, "
                "especially depth_scale and voxel_size");
    }

    total_block_count = std::min(total_block_count,
                                 static_cast<index_t>(hashmap->GetCapacity()));
    block_coordi = block_coordi.Slice(0, 0, total_block_count);
    core::Tensor block_addrs, block_masks;
    hashmap->Activate(block_coordi, block_addrs, block_masks);

    // Customized IndexGet (generic version too slow)
    voxel_block_coords =
            core::Tensor({hashmap->Size(), 3}, core::Int32, device);
    index_t *voxel_block_coord_ptr = voxel_block_coords.GetDataPtr<index_t>();
    bool *block_masks_ptr = block_masks.GetDataPtr<bool>();
    count[0] = 0;
    core::ParallelFor(device, total_block_count,
                      [=] OPEN3D_DEVICE(index_t workload_idx) {
                          if (block_masks_ptr[workload_idx]) {
                              index_t idx = OPEN3D_ATOMIC_ADD(count_ptr, 1);
                              index_t offset_lhs = 3 * idx;
                              index_t offset_rhs = 3 * workload_idx;
                              voxel_block_coord_ptr[offset_lhs + 0] =
                                      block_coordi_ptr[offset_rhs + 0];
                              voxel_block_coord_ptr[offset_lhs + 1] =
                                      block_coordi_ptr[offset_rhs + 1];
                              voxel_block_coord_ptr[offset_lhs + 2] =
                                      block_coordi_ptr[offset_rhs + 2];
                          }
                      });
    OPEN3D_CUDA_CHECK(hipDeviceSynchronize());
}

#define FN_ARGUMENTS                                                      \
    const core::Tensor &depth, const core::Tensor &color,                 \
            const core::Tensor &indices, const core::Tensor &block_keys,  \
            TensorMap &block_values, const core::Tensor &depth_intrinsic, \
            const core::Tensor &color_intrinsic,                          \
            const core::Tensor &extrinsic, index_t resolution,            \
            float voxel_size, float sdf_trunc, float depth_scale,         \
            float depth_max

template void IntegrateCUDA<uint16_t, uint8_t, float, uint16_t, uint16_t>(
        FN_ARGUMENTS);
template void IntegrateCUDA<uint16_t, uint8_t, float, float, float>(
        FN_ARGUMENTS);
template void IntegrateCUDA<float, float, float, uint16_t, uint16_t>(
        FN_ARGUMENTS);
template void IntegrateCUDA<float, float, float, float, float>(FN_ARGUMENTS);

#undef FN_ARGUMENTS

#define FN_ARGUMENTS                                                           \
    std::shared_ptr<core::HashMap> &hashmap, const TensorMap &block_value_map, \
            const core::Tensor &range_map, TensorMap &renderings_map,          \
            const core::Tensor &intrinsic, const core::Tensor &extrinsic,      \
            index_t h, index_t w, index_t block_resolution, float voxel_size,  \
            float depth_scale, float depth_min, float depth_max,               \
            float weight_threshold, float trunc_voxel_multiplier,              \
            int range_map_down_factor

template void RayCastCUDA<float, uint16_t, uint16_t>(FN_ARGUMENTS);
template void RayCastCUDA<float, float, float>(FN_ARGUMENTS);

#undef FN_ARGUMENTS

#define FN_ARGUMENTS                                                           \
    const core::Tensor &block_indices, const core::Tensor &nb_block_indices,   \
            const core::Tensor &nb_block_masks,                                \
            const core::Tensor &block_keys, const TensorMap &block_value_map,  \
            core::Tensor &points, core::Tensor &normals, core::Tensor &colors, \
            index_t block_resolution, float voxel_size,                        \
            float weight_threshold, index_t &valid_size

template void ExtractPointCloudCUDA<float, uint16_t, uint16_t>(FN_ARGUMENTS);
template void ExtractPointCloudCUDA<float, float, float>(FN_ARGUMENTS);

#undef FN_ARGUMENTS

void ExtractTriangleMeshCUDA(const core::Tensor &block_indices,
                             const core::Tensor &inv_block_indices,
                             const core::Tensor &nb_block_indices,
                             const core::Tensor &nb_block_masks,
                             const core::Tensor &block_keys,
                             const std::vector<core::Tensor> &block_values,
                             core::Tensor &vertices,
                             core::Tensor &triangles,
                             core::Tensor &vertex_normals,
                             core::Tensor &vertex_colors,
                             index_t block_resolution,
                             float voxel_size,
                             float weight_threshold,
                             index_t &vertex_count);

#define FN_ARGUMENTS                                                          \
    const core::Tensor &block_indices, const core::Tensor &inv_block_indices, \
            const core::Tensor &nb_block_indices,                             \
            const core::Tensor &nb_block_masks,                               \
            const core::Tensor &block_keys, const TensorMap &block_value_map, \
            core::Tensor &vertices, core::Tensor &triangles,                  \
            core::Tensor &vertex_normals, core::Tensor &vertex_colors,        \
            index_t block_resolution, float voxel_size,                       \
            float weight_threshold, index_t &vertex_count

template void ExtractTriangleMeshCUDA<float, uint16_t, uint16_t>(FN_ARGUMENTS);
template void ExtractTriangleMeshCUDA<float, float, float>(FN_ARGUMENTS);

#undef FN_ARGUMENTS

}  // namespace voxel_grid
}  // namespace kernel
}  // namespace geometry
}  // namespace t
}  // namespace open3d
