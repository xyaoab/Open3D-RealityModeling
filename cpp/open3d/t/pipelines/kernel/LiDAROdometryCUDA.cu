#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Dispatch.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/Tensor.h"
#include "open3d/t/geometry/kernel/GeometryIndexer.h"
#include "open3d/t/geometry/kernel/GeometryMacros.h"
#include "open3d/t/pipelines/kernel/LiDAROdometryImpl.h"
#include "open3d/t/pipelines/kernel/Reduction6x6Impl.cuh"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {
namespace odometry {

using t::geometry::kernel::NDArrayIndexer;
using t::geometry::kernel::TransformIndexer;

__global__ void ComputeLiDAROdometryPointToPlaneCUDAKernel(
        NDArrayIndexer source_vertex_indexer,
        NDArrayIndexer source_mask_indexer,
        NDArrayIndexer target_vertex_indexer,
        NDArrayIndexer target_mask_indexer,
        NDArrayIndexer target_normal_indexer,
        TransformIndexer proj_transform,
        TransformIndexer src2dst_transform,
        LiDARCalibConfig config,
        float* global_sum,
        float depth_diff) {
    // Find correspondence and obtain Jacobian at (x, y)
    // Note the built-in indexer uses (x, y) and (u, v) convention.

    const int kBlockSize = 256;
    __shared__ float local_sum0[kBlockSize];
    __shared__ float local_sum1[kBlockSize];
    __shared__ float local_sum2[kBlockSize];

    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    if (y >= config.height || x >= config.width) return;

    float J[6] = {0}, reduction[21 + 6 + 2];
    float r = 0;
    bool valid = GetJacobianPointToPlane(
            source_vertex_indexer, source_mask_indexer, target_vertex_indexer,
            target_mask_indexer, target_normal_indexer, proj_transform,
            src2dst_transform, config, depth_diff, x, y, J, r);

    // Dump J, r into JtJ and Jtr
    int offset = 0;
    for (int i = 0; i < 6; ++i) {
        for (int j = 0; j <= i; ++j) {
            reduction[offset++] = J[i] * J[j];
        }
    }
    for (int i = 0; i < 6; ++i) {
        reduction[offset++] = J[i] * r;
    }
    reduction[offset++] = r * r;
    reduction[offset++] = valid;

    // Sum reduction: JtJ(21) and Jtr(6)
    for (size_t i = 0; i < 27; i += 3) {
        local_sum0[tid] = valid ? reduction[i + 0] : 0;
        local_sum1[tid] = valid ? reduction[i + 1] : 0;
        local_sum2[tid] = valid ? reduction[i + 2] : 0;
        __syncthreads();

        BlockReduceSum<float, kBlockSize>(tid, local_sum0, local_sum1,
                                          local_sum2);

        if (tid == 0) {
            atomicAdd(&global_sum[i + 0], local_sum0[0]);
            atomicAdd(&global_sum[i + 1], local_sum1[0]);
            atomicAdd(&global_sum[i + 2], local_sum2[0]);
        }
        __syncthreads();
    }

    // Sum reduction: residual(1) and inlier(1)
    {
        local_sum0[tid] = valid ? reduction[27] : 0;
        local_sum1[tid] = valid ? reduction[28] : 0;
        __syncthreads();

        BlockReduceSum<float, kBlockSize>(tid, local_sum0, local_sum1);
        if (tid == 0) {
            atomicAdd(&global_sum[27], local_sum0[0]);
            atomicAdd(&global_sum[28], local_sum1[0]);
        }
        __syncthreads();
    }
}

void ComputeLiDAROdometryPointToPlaneCUDA(
        // source input
        const core::Tensor& source_vertex_map,
        const core::Tensor& source_mask_map,
        // target input
        const core::Tensor& target_vertex_map,
        const core::Tensor& target_mask_map,
        const core::Tensor& target_normal_map,
        // init transformation
        const core::Tensor& init_source_to_target,
        const core::Tensor& sensor_to_lidar,
        // LiDAR calibration
        const LiDARCalibConfig& config,
        // Output linear system result
        core::Tensor& delta,
        float& inlier_residual,
        int& inlier_count,
        // Other params
        float depth_diff) {
    core::Device device = source_vertex_map.GetDevice();

    // Index source data
    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);
    NDArrayIndexer source_mask_indexer(source_mask_map, 2);

    // Index target data
    NDArrayIndexer target_vertex_indexer(target_vertex_map, 2);
    NDArrayIndexer target_mask_indexer(target_mask_map, 2);
    NDArrayIndexer target_normal_indexer(target_normal_map, 2);

    // Wrap transformation
    t::geometry::kernel::TransformIndexer proj_transform(
            core::Tensor::Eye(3, core::Dtype::Float64, core::Device()),
            (sensor_to_lidar.Matmul(init_source_to_target)).Contiguous());

    t::geometry::kernel::TransformIndexer src2dst_transform(
            core::Tensor::Eye(3, core::Dtype::Float64, core::Device()),
            init_source_to_target.Contiguous());

    // Result
    core::Tensor global_sum = core::Tensor::Zeros({29}, core::Float32, device);
    float* global_sum_ptr = global_sum.GetDataPtr<float>();

    // Launcher config
    const int kThreadSize = 16;
    const dim3 blocks((config.width + kThreadSize - 1) / kThreadSize,
                      (config.height + kThreadSize - 1) / kThreadSize);
    const dim3 threads(kThreadSize, kThreadSize);
    ComputeLiDAROdometryPointToPlaneCUDAKernel<<<blocks, threads, 0,
                                                 core::cuda::GetStream()>>>(
            // Input
            source_vertex_indexer, source_mask_indexer, target_vertex_indexer,
            target_mask_indexer, target_normal_indexer,
            // Transform
            proj_transform, src2dst_transform,
            // LiDAR calib LUTs
            config,
            // Output
            global_sum_ptr,
            // Params
            depth_diff);
    core::cuda::Synchronize();

    DecodeAndSolve6x6(global_sum, delta, inlier_residual, inlier_count);
}

}  // namespace odometry
}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
